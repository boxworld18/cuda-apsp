#include "hip/hip_runtime.h"
// PLEASE MODIFY THIS FILE TO IMPLEMENT YOUR SOLUTION

// Brute Force APSP Implementation:

#include "apsp.h"
#include "cuda_utils.h"

constexpr int THREAD_SIZE = 2;
constexpr int BLOCK_SIZE = 64;
constexpr int SM_SIZE = BLOCK_SIZE * BLOCK_SIZE;
constexpr int MAX_LEN = 100001;

namespace {

__global__ void phase1(int n, int phr, int *graph) {
    __shared__ int res[SM_SIZE];

    int thr_x = THREAD_SIZE * threadIdx.x;
    int thr_y = THREAD_SIZE * threadIdx.y;

    int res_id1 = thr_y * BLOCK_SIZE + thr_x;
    int res_id2 = thr_y * BLOCK_SIZE + thr_x + 1;
    int res_id3 = (thr_y + 1) * BLOCK_SIZE + thr_x;
    int res_id4 = (thr_y + 1) * BLOCK_SIZE + thr_x + 1;

    int abs_x = phr * BLOCK_SIZE + thr_x;
    int abs_y = phr * BLOCK_SIZE + thr_y;
    int abs_id1 = abs_y * n + abs_x;
    int abs_id2 = abs_y * n + abs_x + 1;
    int abs_id3 = (abs_y + 1) * n + abs_x;
    int abs_id4 = (abs_y + 1) * n + abs_x + 1;

    int len1, len2, len3, len4;
    res[res_id1] = len1 = (abs_x < n && abs_y < n) ? graph[abs_id1] : MAX_LEN;
    res[res_id2] = len2 = (abs_x + 1 < n && abs_y < n) ? graph[abs_id2] : MAX_LEN;
    res[res_id3] = len3 = (abs_x < n && abs_y + 1 < n) ? graph[abs_id3] : MAX_LEN;
    res[res_id4] = len4 = (abs_x + 1 < n && abs_y + 1 < n) ? graph[abs_id4] : MAX_LEN;

    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++) {
        int yk1 = res[thr_y * BLOCK_SIZE + k];
        int yk2 = res[(thr_y + 1) * BLOCK_SIZE + k];
        int xk1 = res[k * BLOCK_SIZE + thr_x];
        int xk2 = res[k * BLOCK_SIZE + thr_x + 1];
        len1 = min(len1, yk1 + xk1);
        len2 = min(len2, yk1 + xk2);
        len3 = min(len3, yk2 + xk1);
        len4 = min(len4, yk2 + xk2);
        __syncthreads();
        res[res_id1] = len1;
        res[res_id2] = len2;
        res[res_id3] = len3;
        res[res_id4] = len4;
        __syncthreads();
    }

    if (abs_x + 1 < n) {
        if (abs_y + 1 < n) {
            graph[abs_id1] = len1;
            graph[abs_id2] = len2;
            graph[abs_id3] = len3;
            graph[abs_id4] = len4;
        } else if (abs_y < n) {
            graph[abs_id1] = len1;
            graph[abs_id2] = len2;
        }
    } else if (abs_x < n) {
        if (abs_y + 1 < n) {
            graph[abs_id1] = len1;
            graph[abs_id3] = len3;
        } else if (abs_y < n) {
            graph[abs_id1] = len1;
        }
    }
}

__global__ void phase2(int n, int phr, int *graph) {
    if (blockIdx.x == phr) return;
    
    __shared__ int res[SM_SIZE];
    __shared__ int cen[SM_SIZE];
    
    int thr_x = THREAD_SIZE * threadIdx.x;
    int thr_y = THREAD_SIZE * threadIdx.y;

    int res_id1 = thr_y * BLOCK_SIZE + thr_x;
    int res_id2 = thr_y * BLOCK_SIZE + thr_x + 1;
    int res_id3 = (thr_y + 1) * BLOCK_SIZE + thr_x;
    int res_id4 = (thr_y + 1) * BLOCK_SIZE + thr_x + 1;

    // get center block
    int cen_x = phr * BLOCK_SIZE + thr_x;
    int cen_y = phr * BLOCK_SIZE + thr_y;
    int cen_id1 = cen_y * n + cen_x;
    int cen_id2 = cen_y * n + cen_x + 1;
    int cen_id3 = (cen_y + 1) * n + cen_x;
    int cen_id4 = (cen_y + 1) * n + cen_x + 1;

    cen[res_id1] = (cen_x < n && cen_y < n) ? graph[cen_id1] : MAX_LEN;
    cen[res_id2] = (cen_x + 1 < n && cen_y < n) ? graph[cen_id2] : MAX_LEN;
    cen[res_id3] = (cen_x < n && cen_y + 1 < n) ? graph[cen_id3] : MAX_LEN;
    cen[res_id4] = (cen_x + 1 < n && cen_y + 1 < n) ? graph[cen_id4] : MAX_LEN;

    // get own block
    int abs_x, abs_y;
    if (blockIdx.y) {
        abs_x = blockIdx.x * BLOCK_SIZE + thr_x;
        abs_y = phr * BLOCK_SIZE + thr_y;
    } else {
        abs_x = phr * BLOCK_SIZE + thr_x;
        abs_y = blockIdx.x * BLOCK_SIZE + thr_y;
    }

    int abs_id1 = abs_y * n + abs_x;
    int abs_id2 = abs_y * n + abs_x + 1;
    int abs_id3 = (abs_y + 1) * n + abs_x;
    int abs_id4 = (abs_y + 1) * n + abs_x + 1;

    int len1, len2, len3, len4;
    res[res_id1] = len1 = (abs_x < n && abs_y < n) ? graph[abs_id1] : MAX_LEN;
    res[res_id2] = len2 = (abs_x + 1 < n && abs_y < n) ? graph[abs_id2] : MAX_LEN;
    res[res_id3] = len3 = (abs_x < n && abs_y + 1 < n) ? graph[abs_id3] : MAX_LEN;
    res[res_id4] = len4 = (abs_x + 1 < n && abs_y + 1 < n) ? graph[abs_id4] : MAX_LEN;
    __syncthreads();

    if (blockIdx.y) {
        for (int k = 0; k < BLOCK_SIZE; k++) {
            int path1 = cen[thr_y * BLOCK_SIZE + k] + res[k * BLOCK_SIZE + thr_x];
            int path2 = cen[thr_y * BLOCK_SIZE + k] + res[k * BLOCK_SIZE + thr_x + 1];
            int path3 = cen[(thr_y + 1) * BLOCK_SIZE + k] + res[k * BLOCK_SIZE + thr_x];
            int path4 = cen[(thr_y + 1) * BLOCK_SIZE + k] + res[k * BLOCK_SIZE + thr_x + 1];
            len1 = min(len1, path1);
            len2 = min(len2, path2);
            len3 = min(len3, path3);
            len4 = min(len4, path4);
            __syncthreads();
            res[res_id1] = len1;
            res[res_id2] = len2;
            res[res_id3] = len3;
            res[res_id4] = len4;
            __syncthreads();
        }
    } else {
        for (int k = 0; k < BLOCK_SIZE; k++) {
            int path1 = res[thr_y * BLOCK_SIZE + k] + cen[k * BLOCK_SIZE + thr_x];
            int path2 = res[thr_y * BLOCK_SIZE + k] + cen[k * BLOCK_SIZE + thr_x + 1];
            int path3 = res[(thr_y + 1) * BLOCK_SIZE + k] + cen[k * BLOCK_SIZE + thr_x];
            int path4 = res[(thr_y + 1) * BLOCK_SIZE + k] + cen[k * BLOCK_SIZE + thr_x + 1];
            len1 = min(len1, path1);
            len2 = min(len2, path2);
            len3 = min(len3, path3);
            len4 = min(len4, path4);
            __syncthreads();
            res[res_id1] = len1;
            res[res_id2] = len2;
            res[res_id3] = len3;
            res[res_id4] = len4;
            __syncthreads();
        }
    }

    if (abs_x + 1 < n) {
        if (abs_y + 1 < n) {
            graph[abs_id1] = len1;
            graph[abs_id2] = len2;
            graph[abs_id3] = len3;
            graph[abs_id4] = len4;
        } else if (abs_y < n) {
            graph[abs_id1] = len1;
            graph[abs_id2] = len2;
        }
    } else if (abs_x < n) {
        if (abs_y + 1 < n) {
            graph[abs_id1] = len1;
            graph[abs_id3] = len3;
        } else if (abs_y < n) {
            graph[abs_id1] = len1;
        }
    }
}

__global__ void phase3(int n, int phr, int *graph) {
    if (blockIdx.x == phr || blockIdx.y == phr) return;
    
    __shared__ int ver[SM_SIZE];
    __shared__ int hor[SM_SIZE];

    int thr_x = THREAD_SIZE * threadIdx.x;
    int thr_y = THREAD_SIZE * threadIdx.y;

    int res_id1 = thr_y * BLOCK_SIZE + thr_x;
    int res_id2 = thr_y * BLOCK_SIZE + thr_x + 1;
    int res_id3 = (thr_y + 1) * BLOCK_SIZE + thr_x;
    int res_id4 = (thr_y + 1) * BLOCK_SIZE + thr_x + 1;

    // get horizontal block
    int hor_x = phr * BLOCK_SIZE + thr_x;
    int hor_y = blockIdx.y * BLOCK_SIZE + thr_y;
    int hor_id1 = hor_y * n + hor_x;
    int hor_id2 = hor_y * n + hor_x + 1;
    int hor_id3 = (hor_y + 1) * n + hor_x;
    int hor_id4 = (hor_y + 1) * n + hor_x + 1;

    hor[res_id1] = (hor_x < n && hor_y < n) ? graph[hor_id1] : MAX_LEN;
    hor[res_id2] = (hor_x + 1 < n && hor_y < n) ? graph[hor_id2] : MAX_LEN;
    hor[res_id3] = (hor_x < n && hor_y + 1 < n) ? graph[hor_id3] : MAX_LEN;
    hor[res_id4] = (hor_x + 1 < n && hor_y + 1 < n) ? graph[hor_id4] : MAX_LEN;

    // get vertical block
    int ver_x = blockIdx.x * BLOCK_SIZE + thr_x;
    int ver_y = phr * BLOCK_SIZE + thr_y;
    int ver_id1 = ver_y * n + ver_x;
    int ver_id2 = ver_y * n + ver_x + 1;
    int ver_id3 = (ver_y + 1) * n + ver_x;
    int ver_id4 = (ver_y + 1) * n + ver_x + 1;

    ver[res_id1] = (ver_x < n && ver_y < n) ? graph[ver_id1] : MAX_LEN;
    ver[res_id2] = (ver_x + 1 < n && ver_y < n) ? graph[ver_id2] : MAX_LEN;
    ver[res_id3] = (ver_x < n && ver_y + 1 < n) ? graph[ver_id3] : MAX_LEN;
    ver[res_id4] = (ver_x + 1 < n && ver_y + 1 < n) ? graph[ver_id4] : MAX_LEN;

    // get own block
    int abs_x = blockIdx.x * BLOCK_SIZE + thr_x;
    int abs_y = blockIdx.y * BLOCK_SIZE + thr_y;
    int abs_id1 = abs_y * n + abs_x;
    int abs_id2 = abs_y * n + abs_x + 1;
    int abs_id3 = (abs_y + 1) * n + abs_x;
    int abs_id4 = (abs_y + 1) * n + abs_x + 1;

    int len1, len2, len3, len4;
    len1 = (abs_x < n && abs_y < n) ? graph[abs_id1] : MAX_LEN;
    len2 = (abs_x + 1 < n && abs_y < n) ? graph[abs_id2] : MAX_LEN;
    len3 = (abs_x < n && abs_y + 1 < n) ? graph[abs_id3] : MAX_LEN;
    len4 = (abs_x + 1 < n && abs_y + 1 < n) ? graph[abs_id4] : MAX_LEN;

    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++) {
        int path1 = hor[thr_y * BLOCK_SIZE + k] + ver[k * BLOCK_SIZE + thr_x];
        int path2 = hor[thr_y * BLOCK_SIZE + k] + ver[k * BLOCK_SIZE + thr_x + 1];
        int path3 = hor[(thr_y + 1) * BLOCK_SIZE + k] + ver[k * BLOCK_SIZE + thr_x];
        int path4 = hor[(thr_y + 1) * BLOCK_SIZE + k] + ver[k * BLOCK_SIZE + thr_x + 1];
        len1 = min(len1, path1);
        len2 = min(len2, path2);
        len3 = min(len3, path3);
        len4 = min(len4, path4);
    }

    if (abs_x + 1 < n) {
        if (abs_y + 1 < n) {
            graph[abs_id1] = len1;
            graph[abs_id2] = len2;
            graph[abs_id3] = len3;
            graph[abs_id4] = len4;
        } else if (abs_y < n) {
            graph[abs_id1] = len1;
            graph[abs_id2] = len2;
        }
    } else if (abs_x < n) {
        if (abs_y + 1 < n) {
            graph[abs_id1] = len1;
            graph[abs_id3] = len3;
        } else if (abs_y < n) {
            graph[abs_id1] = len1;
        }
    }

}

}

void apsp(int n, /* device */ int *graph) {
    int num_blocks = (n - 1) / BLOCK_SIZE + 1;
    for (int phr = 0; phr < num_blocks; phr++) {
        dim3 thr(BLOCK_SIZE / THREAD_SIZE, BLOCK_SIZE / THREAD_SIZE);
        dim3 blk1(1, 1);
        dim3 blk2(num_blocks, 2);
        dim3 blk3(num_blocks, num_blocks + 1);

        phase1<<<blk1, thr>>>(n, phr, graph);
        phase2<<<blk2, thr>>>(n, phr, graph);
        phase3<<<blk3, thr>>>(n, phr, graph);
    }
}